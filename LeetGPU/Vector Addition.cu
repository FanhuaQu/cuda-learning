#include <hip/hip_runtime.h>

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int block_idx = blockIdx.x;
    int thread_idx = threadIdx.x;
    // offset of each block
    int i = block_idx * 256 + thread_idx;
    if(i < N){
        C[i] = A[i] + B[i];
    }
    
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // copy from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
    

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // launch kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // copy from device to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
