#include <hip/hip_runtime.h>

__global__ void count_equal_kernel(const int* input, int* output, int N, int K) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid >= N)
        return;
    if(input[tid] == K){
        atomicAdd(output, 1);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const int* input, int* output, int N, int K) {
    int *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, sizeof(int));

    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    count_equal_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N, K);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
