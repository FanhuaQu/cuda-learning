#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < N){
        output[tid] = input[tid] > 0 ? input[tid] : input[tid] * 0.01;
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int N) {
    float *d_input, *d_output;

    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
}