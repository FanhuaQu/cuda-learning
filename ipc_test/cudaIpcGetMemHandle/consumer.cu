// consumer.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fcntl.h>
#include <unistd.h>

int main() {
    hipSetDevice(1);

    hipIpcMemHandle_t handle;

    // 从文件读取句柄
    int fd = open("memhandle.bin", O_RDONLY);
    read(fd, &handle, sizeof(handle));
    close(fd);

    int canAccessPeer = 0;
    if(hipSuccess != hipDeviceCanAccessPeer(&canAccessPeer, 1, 0)){
        printf("hipDeviceCanAccessPeer failed\n");
    }
    else{
        printf("hipDeviceCanAccessPeer success, canAccessPeer=%d\n", canAccessPeer);
    }

    int *d_data;
    hipError_t err = hipIpcOpenMemHandle((void**)&d_data, handle, hipIpcMemLazyEnablePeerAccess);
    if (err != hipSuccess) {
        fprintf(stderr, "hipIpcOpenMemHandle failed: %s\n", hipGetErrorString(err));
        return 1;
    }
    else{
        printf("hipIpcOpenMemHandle success, canAccessPeer=%d\n", canAccessPeer);
    }
    printf("canAccessPeer=%d\n", canAccessPeer);

    // 读取数据
    size_t N = 10;
    int h_data[10] = {0};
    hipMemcpy(h_data, d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("进程 B: 从共享显存读取到数据：\n");
    for (size_t i = 0; i < N; i++) {
        printf("%d ", h_data[i]);
    }
    printf("\n");

    hipIpcCloseMemHandle(d_data);
    return 0;
}
